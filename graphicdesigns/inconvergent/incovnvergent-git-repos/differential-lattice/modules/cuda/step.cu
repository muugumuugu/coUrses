#include "hip/hip_runtime.h"
#define THREADS _THREADS_
#define PROX _PROX_

__device__ float dist(const float *a, const float *b, const int ii, const int jj){
    return sqrt(powf(a[ii]-b[jj], 2.0f)+powf(a[ii+1]-b[jj+1], 2.0f));
}

__device__ int get_candidates(
  const int nz,
  const int zi,
  const int zj,
  const int *zone_num,
  const int *zone_node,
  const int zone_leap,
  const float *xy,
  const float outer_influence_rad,
  const int ii,
  int *proximity
){

  int zk;
  int jj;
  float dd;

  int count = 0;

  for (int a=max(zi-1,0);a<min(zi+2,nz);a++){
    for (int b=max(zj-1,0);b<min(zj+2,nz);b++){
      zk = a*nz+b;
      for (int k=0;k<zone_num[zk];k++){
        jj = 2*zone_node[zk*zone_leap+k];
        dd = dist(xy, xy, ii, jj);
        if (dd<outer_influence_rad && dd>0.0f){
          proximity[count] = jj/2;
          count += 1;
        }
      }
    }
  }

  return count;
}

__global__ void step(
  const int n,
  const int nz,
  const int zone_leap,
  const float *xy,
  float *dxy,
  int *tmp,
  int *links,
  int *link_counts,
  const int *zone_num,
  const int *zone_node,
  const float stp,
  const float reject_stp,
  const float spring_stp,
  const float cohesion_stp,
  const float spring_reject_rad,
  const float spring_attract_rad,
  const int max_capacity,
  const float outer_influence_rad,
  const float link_ignore_rad
){
  const int i = blockIdx.x*THREADS + threadIdx.x;

  if (i>=n){
    return;
  }

  const int ii = 2*i;
  const int zi = (int) floor(xy[ii]*nz);
  const int zj = (int) floor(xy[ii+1]*nz);

  float sx = 0.0f;
  float sy = 0.0f;
  float dx = 0.0f;
  float dy = 0.0f;
  float dd = 0.0f;

  float mx = 0.0f;
  float my = 0.0f;
  float mm = 0.0f;


  int jj;
  int aa;

  int link_count = 0;

  int proximity[PROX];
  int cand_count = get_candidates(nz, zi, zj, zone_num, zone_node, zone_leap,
      xy, outer_influence_rad, ii, proximity);

  bool linked;

  for (int k=0;k<cand_count;k++){

    jj = 2*proximity[k];

    dx = xy[ii] - xy[jj];
    dy = xy[ii+1] - xy[jj+1];
    dd = sqrt(powf(dx, 2.0f) + powf(dy, 2.0f));

    if (dd<=0.0f){
      continue;
    }

    linked = true;
    for (int l=0;l<cand_count;l++){
      aa = 2*proximity[l];
      if (dd>link_ignore_rad){
        linked = false;
        break;
      }
      if (dd>max(dist(xy, xy, aa, ii), dist(xy, xy, jj, aa))){
        linked = false;
        break;
      }
    }

    dx /= dd;
    dy /= dd;

    mx += xy[jj];
    my += xy[jj+1];

    if (linked){
      links[10*i+link_count] = jj/2;
      link_count += 1;
      if (dd>spring_attract_rad){
        sx += -dx*spring_stp;
        sy += -dy*spring_stp;
      }
      else if(dd<spring_reject_rad){
        sx += dx*spring_stp;
        sy += dy*spring_stp;
      }
    }
    else{ // unlinked
      sx += dx*reject_stp;
      sy += dy*reject_stp;
    }
  }

  mx = mx/(float)cand_count - xy[ii];
  my = my/(float)cand_count - xy[ii+1];
  mm = sqrt(mx*mx + my*my);

  mx *= -cohesion_stp/mm;
  my *= -cohesion_stp/mm;

  dxy[ii] = (sx+mx)*stp;
  dxy[ii+1] = (sy+my)*stp;
  tmp[i] = cand_count;
  link_counts[i] = link_count;

}

