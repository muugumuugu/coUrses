#include "hip/hip_runtime.h"
#define THREADS _THREADS_
#define PI 3.141592654f

__global__ void circle(const int n,
                       const float *rnd,
                       float *xy,
                       const float rad,
                       const float *mid,
                       const int grains) {

  const int i = blockIdx.x*THREADS + threadIdx.x;

  if (i >= n) {
    return;
  }

  const int ii = 2*i;
  const int iii = 3*i;
  const int k = 2*(int)floor((float)i/(float)grains);

  const float t = 2 * PI * rnd[iii];
  const float u = rnd[iii+1] + rnd[iii+2];

  float r;
  if (u>1.0f){
    r = rad*(2.0f-u);
  } else {
    r = rad*u;
  }

  xy[ii] = mid[k] + r * cos(t);
  xy[ii+1] = mid[k+1] + r *sin(t);

}

