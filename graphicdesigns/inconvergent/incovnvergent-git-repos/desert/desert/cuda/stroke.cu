#include "hip/hip_runtime.h"
#define THREADS _THREADS_

__global__ void stroke(const int n,
                       const float *ab,
                       const float *rnd,
                       float *xy,
                       const int grains) {

  const int i = blockIdx.x*THREADS + threadIdx.x;
  const int ii = 2*i;

  if (i >= n) {
    return;
  }

  const int k = 4*(int)floor((float)i/(float)grains);

  const float dx = ab[k+2] - ab[k];
  const float dy = ab[k+3] - ab[k+1];

  const float r = rnd[i];

  xy[ii] = ab[k] + r*dx;
  xy[ii+1] = ab[k+1] + r*dy;

}

